#include "hip/hip_runtime.h"
/*

 The MIT License

 Copyright (c) 1997-2010 Center for the Simulation of Accidental Fires and 
 Explosions (CSAFE), and  Scientific Computing and Imaging Institute (SCI), 
 University of Utah.

 License for the specific language governing rights and limitations under
 Permission is hereby granted, free of charge, to any person obtaining a 
 copy of this software and associated documentation files (the "Software"),
 to deal in the Software without restriction, including without limitation 
 the rights to use, copy, modify, merge, publish, distribute, sublicense, 
 and/or sell copies of the Software, and to permit persons to whom the 
 Software is furnished to do so, subject to the following conditions:

 The above copyright notice and this permission notice shall be included 
 in all copies or substantial portions of the Software.

 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS 
 OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL 
 THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER 
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING 
 FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER 
 DEALINGS IN THE SOFTWARE.

 */

#include <CCA/Components/Examples/PoissonGPU1.h>
#include <CCA/Components/Examples/ExamplesLabel.h>
#include <Core/ProblemSpec/ProblemSpec.h>
#include <Core/Grid/Variables/NCVariable.h>
#include <Core/Grid/Variables/NodeIterator.h>
#include <Core/Grid/SimulationState.h>
#include <Core/Grid/CUDATask.h>
#include <Core/Grid/Level.h>
#include <Core/Grid/SimpleMaterial.h>
#include <Core/Grid/Variables/VarTypes.h>
#include <Core/Parallel/ProcessorGroup.h>
#include <CCA/Ports/Scheduler.h>
#include <Core/Malloc/Allocator.h>
#include <Core/Grid/BoundaryConditions/BCDataArray.h>
#include <Core/Grid/BoundaryConditions/BoundCond.h>
#include <CCA/Components/Schedulers/CUDADevice.h>

#include <sci_defs/cuda_defs.h>

using namespace Uintah;

PoissonGPU1::PoissonGPU1(const ProcessorGroup* myworld) :
UintahParallelComponent(myworld)
{

  phi_label = VarLabel::create("phi", NCVariable<double>::getTypeDescription());
  residual_label = VarLabel::create("residual",
      sum_vartype::getTypeDescription());
}

PoissonGPU1::~PoissonGPU1()
{
  VarLabel::destroy( phi_label);
  VarLabel::destroy( residual_label);
}
//______________________________________________________________________
//
void
PoissonGPU1::problemSetup(const ProblemSpecP& params,
    const ProblemSpecP& restart_prob_spec, GridP& /*grid*/,
    SimulationStateP& sharedState)
{
  sharedState_ = sharedState;
  ProblemSpecP poisson = params->findBlock("Poisson");

  poisson->require("delt", delt_);

  mymat_ = scinew SimpleMaterial();

  sharedState->registerSimpleMaterial(mymat_);
}
//______________________________________________________________________
//
void
PoissonGPU1::scheduleInitialize(const LevelP& level, SchedulerP& sched)
{
  Task* task = scinew Task("PoissonGPU1::initialize",
      this, &PoissonGPU1::initialize);

  task->computes(phi_label);
  task->computes(residual_label);
  sched->addTask(task, level->eachPatch(), sharedState_->allMaterials());
}
//______________________________________________________________________
//
void
PoissonGPU1::scheduleComputeStableTimestep(const LevelP& level,
    SchedulerP& sched)
{
  Task* task = scinew Task("PoissonGPU1::computeStableTimestep",
      this, &PoissonGPU1::computeStableTimestep);

  task->requires(Task::NewDW, residual_label);
  task->computes(sharedState_->get_delt_label(), level.get_rep());
  sched->addTask(task, level->eachPatch(), sharedState_->allMaterials());
}
//______________________________________________________________________
//
void
PoissonGPU1::scheduleTimeAdvance(const LevelP& level, SchedulerP& sched)
{
  Task* task = scinew Task("PoissonGPU1::timeAdvance",
      this, &PoissonGPU1::timeAdvance);
//  CUDATask* task = scinew CUDATask("PoissonGPU1::timeAdvance",
//      this, &PoissonGPU1::timeAdvance);

  task->requires(Task::OldDW, phi_label, Ghost::AroundNodes, 1);
  task->computes(phi_label);
  task->computes(residual_label);
  sched->addTask(task, level->eachPatch(), sharedState_->allMaterials());
}
//______________________________________________________________________
//
void
PoissonGPU1::computeStableTimestep(const ProcessorGroup* pg,
    const PatchSubset* patches, const MaterialSubset* /*matls*/,
    DataWarehouse*, DataWarehouse* new_dw)
{

  if (pg->myrank() == 0)
  {
    sum_vartype residual;
    new_dw->get(residual, residual_label);
    cerr << "Residual=" << residual << '\n';
  }
  new_dw->put(delt_vartype(delt_), sharedState_->get_delt_label(), getLevel(patches));
}
//______________________________________________________________________
//
void
PoissonGPU1::initialize(const ProcessorGroup*, const PatchSubset* patches,
    const MaterialSubset* matls, DataWarehouse* /*old_dw*/,
    DataWarehouse* new_dw)
{
  int matl = 0;
  for (int p = 0; p < patches->size(); p++)
  {
    const Patch* patch = patches->get(p);

    NCVariable<double> phi;
    new_dw->allocateAndPut(phi, phi_label, matl, patch);
    phi.initialize(0.0);

    for (Patch::FaceType face = Patch::startFace; face <= Patch::endFace; face
        = Patch::nextFace(face))
    {
      if (patch->getBCType(face) == Patch::None)
      {
        int numChildren = patch->getBCDataArray(face)->getNumberChildren(
            matl);
        for (int child = 0; child < numChildren; child++)
        {
          Iterator nbound_ptr, nu;

          const BoundCondBase* bcb = patch->getArrayBCValues(face, matl, "Phi", nu, nbound_ptr, child);
          const BoundCond<double>* bc = dynamic_cast<const BoundCond<double>*> (bcb);
          double value = bc->getValue();
          for (nbound_ptr.reset(); !nbound_ptr.done(); nbound_ptr++)
          {
            phi[*nbound_ptr] = value;
          }
          delete bcb;
        }
      }
    }

    new_dw->put(sum_vartype(-1), residual_label);
  }
}

//______________________________________________________________________
//
// @brief A kernel that applies the stencil used in timeAdvance(...)
// @param domainSize a three component vector that gives the size of the domain as (x,y,z)
// @param domainLower a three component vector that gives the lower corner of the workarea as (x,y,z)
// @param residual the residual calculated by this individual kernel 
// @param oldphi pointer to the source phi allocated on the device
// @param newphi pointer to the sink phi allocated on the device
__global__ void timeAdvanceKernel(uint3 domainSize, uint3 domainLower, double *phi, double *newphi,
    double *residual) {

//  __shared__ double[] residual_device;
  // calculate the indices
//  int indxX = domainLower.x + blockDim.x * blockIdx.x + threadIdx.x;
//  int indxY = domainLower.y + blockDim.y * blockIdx.y + threadIdx.y;
//  int indxZ = domainLower.z + blockDim.z * blockIdx.z + threadIdx.z;
  int tidX = blockDim.x * blockIdx.x + threadIdx.x;
  int tidY = blockDim.y * blockIdx.y + threadIdx.y;
  int tidZ = blockDim.z * blockIdx.z + threadIdx.z;

  int dz = domainSize.x - 1;
  int dy = domainSize.y - 1;
  int dx = domainSize.z - 1;
  int offset = 1;

// calculate the offset in the dw representation
  int tid = tidX + ((dx + offset) * tidY + (dx + offset) * (dy + offset) * tidZ);

  newphi[tid] = (1.0 / 6.0)
      * (phi[tid - offset] + phi[tid + offset] + phi[tid - (dx + offset)]
          + phi[tid + dx + offset] + phi[tid - (dx + offset) * (offset + dy)]
          + phi[tid + (offset + dx) * (offset + dy)]);

  double diff = newphi[tid] - phi[tid];
  // this will cause a race condition. what we need is a scan to compute this
  // in conjunction with atomicAdd() and __shared__ double[] residual_device;
  *residual += diff * diff;
}

//______________________________________________________________________
//
void
PoissonGPU1::timeAdvance(const ProcessorGroup*,
                         const PatchSubset* patches,
                         const MaterialSubset* matls,
                         DataWarehouse* old_dw,
                         DataWarehouse* new_dw) //,
//                         int deviceID = 0,
//                         CUDADevice *deviceProperties = NULL)
{
  //
  int matl = 0;
  int previousPatchSize = 0;// this is to see if we need to release and reallocate between computations
  int size = 0;

  // declare device and host memory
  double* newphi_device;
  double* phi_device;
  double* phi_host;
  double* newphi_host;

  // find the "best" device for hipSetDevice()
  int num_devices, device;
  hipGetDeviceCount(&num_devices);
  if (num_devices > 1) {
    int max_multiprocessors = 0, max_device = 0;
    for (device = 0; device < num_devices; device++) {
      hipDeviceProp_t properties;
      hipGetDeviceProperties(&properties, device);
      if (max_multiprocessors < properties.multiProcessorCount) {
        max_multiprocessors = properties.multiProcessorCount;
        max_device = device;
      }
    }
    hipSetDevice(max_device);
  }

  // Do time steps
  for (int p = 0; p < patches->size(); p++)
  {
    const Patch* patch = patches->get(p);
    constNCVariable<double> phi;
    old_dw->get(phi, phi_label, matl, patch, Ghost::AroundNodes, 1);

    NCVariable<double> newphi;
    new_dw->allocateAndPut(newphi, phi_label, matl, patch);
    newphi.copyPatch(phi, newphi.getLowIndex(), newphi.getHighIndex());

    double residual = 0;
    IntVector l = patch->getNodeLowIndex();
    IntVector h = patch->getNodeHighIndex();
    IntVector s = h-l;
    int xdim = s.x(), ydim = s.y(), zdim = s.z();
    size = xdim * ydim * zdim * sizeof(double);

    l += IntVector(patch->getBCType(Patch::xminus) == Patch::Neighbor?0:1,
        patch->getBCType(Patch::yminus) == Patch::Neighbor?0:1,
        patch->getBCType(Patch::zminus) == Patch::Neighbor?0:1);
    h -= IntVector(patch->getBCType(Patch::xplus) == Patch::Neighbor?0:1,
        patch->getBCType(Patch::yplus) == Patch::Neighbor?0:1,
        patch->getBCType(Patch::zplus) == Patch::Neighbor?0:1);

    // check if we need to reallocate
    if (size != previousPatchSize)
    {
      if (previousPatchSize != 0)
      {
        hipFree(phi_device);
        hipFree(newphi_device);
      }
      hipMalloc(&phi_device, size);
      hipMalloc(&newphi_device, size);
    }

    //__________________________________
    //  Memory Allocation
    phi_host = (double*) phi.getWindow()->getData()->getPointer();
    newphi_host = (double*) newphi.getWindow()->getData()->getPointer();

    // allocate space on the device
    // TODO
    // Fix this so when we have >= CCv2.0 we can use pinned host mem for phi
    hipMemcpy(phi_device, phi_host, size, hipMemcpyHostToDevice);
    hipMemcpy(newphi_device, newphi_host, size, hipMemcpyHostToDevice);

    uint3 domainSize = make_uint3(xdim, ydim, zdim);
    uint3 domainLower = make_uint3(l.x(), l.y(), l.z());
    int totalBlocks = size / (sizeof(double) * xdim * ydim * zdim);
    dim3 threadsPerBlock(xdim, ydim, zdim);

    if (size % (totalBlocks) != 0) {
      totalBlocks++;
    }

    // launch kernel
    timeAdvanceKernel<<< totalBlocks, threadsPerBlock >>>(domainSize, domainLower, phi_device, newphi_device, &residual);

    hipDeviceSynchronize();
    hipMemcpy(newphi_host, newphi_device, size, hipMemcpyDeviceToHost);

    // now store residual that was device calculated
    new_dw->put(sum_vartype(residual), residual_label);

    //    //__________________________________
    //    //  3D-Pointer Stencil operation for reference
    //    double*** phi_data = (double***) phi.getWindow()->getData()->get3DPointer();
    //    double*** newphi_data = (double***) newphi.getWindow()->getData()->get3DPointer();
    //    double diff;
    //
    //    int zlen = s.z()-1;
    //    int ylen = s.y()-1;
    //    int xlen = s.x()-1;
    //    for (int i = 1; i < zlen; i++) {
    //      for (int j = 1; j < ylen; j++) {
    //        for (int k = 1; k < xlen; k++) {
    //
    //          double xminus = phi_data[i-1][j][k];
    //          double xplus  = phi_data[i+1][j][k];
    //          double yminus = phi_data[i][j-1][k];
    //          double yplus  = phi_data[i][j+1][k];
    //          double zminus = phi_data[i][j][k-1];
    //          double zplus  = phi_data[i][j][k+1];
    //
    //          newphi_data[i][j][k] = (1./6) * (xminus + xplus + yminus + yplus  + zminus + zplus);
    //
    //          diff = newphi_data[i][j][k] - phi_data[i][j][k];
    //          residual += diff * diff;
    //        }
    //      }
    //    }
    //    new_dw->put(sum_vartype(residual), residual_label);
  }

  // free up allocated memory
  hipFree(phi_device);
  hipFree(newphi_device);
}
