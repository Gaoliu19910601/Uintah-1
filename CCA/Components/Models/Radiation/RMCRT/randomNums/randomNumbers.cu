#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <MersenneTwister.h>

#define BLKWIDTH 32

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)


//______________________________________________________________________
//
//
//  The following compares the random number generation on the CPU vs GPU
//
//
//______________________________________________________________________



//______________________________________________________________________
//
inline int RoundDown(double d)
{
   if(d<0){
    int i=-(int)-d;
    if((double)i == d)
      return i;
    else
      return i-1;
  } else {
    return (int)d;
  }
}
//______________________________________________________________________
//
inline int RoundUp(double d)
{
    if(d>=0){
        if((d-(int)d) == 0)
            return (int)d;
        else
            return (int)(d+1);
    } else {
        return (int)d;
    }
}
//______________________________________________________________________
//
void stopwatch( char message[], time_t start)
 
{    
    double secs;
    time_t stop;                 /* timing variables             */
            
    stop = time(NULL);
    secs = difftime(stop, start);               
    fprintf(stdout,"    %.f [s] %s  \n",secs, message);       
 }
//______________________________________________________________________
//  CPU based random number generations
void randCPU( double *M, int nRandNums)
{
  MTRand mTwister;
  for (int i = 0; i< nRandNums; i++){
    M[i] = mTwister.rand();
    // printf( "%i rand: %g \n",i, M[i]);
  }
}


//______________________________________________________________________
//  Determine device properties
void deviceProperties( int &maxThreadsPerBlock )
{  
  // Number of CUDA devices
  int devCount;
  hipGetDeviceCount(&devCount);

  // Iterate through devices
  for (int deviceNum = 0; deviceNum < devCount; ++deviceNum){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceNum);
   // printDevProp(deviceProp);

    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
  }
}

//______________________________________________________________________
//  This is the host side random number generation using cuda
void randHostGPU( double *M, int nRandNums)
{
  int size = nRandNums* sizeof(double);
  double* Md;

  //__________________________________
  //  allocate device memory and copy memory to the device
  hipMalloc( (void**)&Md, size);  
  
  hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
  
  //__________________________________
  // Create pseudo-random number generator
  // set the seed 
  // generate the numbers
  hiprandGenerator_t randGen;
  
  hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);

  hiprandSetPseudoRandomGeneratorSeed(randGen, 1234ULL);

  hiprandGenerateUniformDouble(randGen, Md, nRandNums);
 
  
  //__________________________________
  //   copy from device memory and free device matrices
  hipMemcpy( M, Md, size, hipMemcpyDeviceToHost );
  hipFree( Md );
  hiprandDestroyGenerator(randGen);
}

//______________________________________________________________________
//    Returns an random number
__device__ double randDevice(hiprandState* globalState)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState localState = globalState[tid];
  double val = hiprand(&localState);
  globalState[tid] = localState;
  return (double)val * (1.0/4294967295.0);
}


//______________________________________________________________________
//    Returns an random number  
__device__ double randDblExcDevice(hiprandState* globalState)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState localState = globalState[tid];
    
    double val = hiprand_uniform_double(&localState);
    
    globalState[tid] = localState;
    return ( (double)val + 0.5 ) * (1.0/4294967296.0);
}

//______________________________________________________________________
//    Kernel:  
__global__ void randNumKernel( hiprandState* randNumStates, double* M, double* N, int nRandNums )
{

  int tx  = threadIdx.x;
  int ty  = threadIdx.y;
  int row = blockIdx.y * BLKWIDTH + tx;
  int col = blockIdx.x * BLKWIDTH + ty;
  int c   = row * nRandNums +col;
  
  for (int k = 0; k < nRandNums; ++k){
    M[k] = randDblExcDevice( randNumStates );
    N[k] = randDevice( randNumStates );
  }
}

//______________________________________________________________________
//  Device side random number generator
void randDeviceGPU( double *M, double *N,int nRandNums)
{
  int size = nRandNums* sizeof(double);
  double* Md;
  double* Nd;
  //__________________________________
  //  allocate device memory and copy memory to the device
  hipMalloc( (void**)&Md, size);  
  hipMalloc( (void**)&Nd, size);
  //__________________________________
  //  copy host memory -> device
  hipMemcpy( Md, M, size, hipMemcpyHostToDevice );
  hipMemcpy( Nd, N, size, hipMemcpyHostToDevice );  
  //__________________________________
  //
  int maxThreadsPerBlock = 0;
  deviceProperties( maxThreadsPerBlock );
  
  int xMaxThreadsPerBlock = BLKWIDTH;
  int yMaxThreadsPerBlock = BLKWIDTH;
  maxThreadsPerBlock = xMaxThreadsPerBlock * yMaxThreadsPerBlock;       // hardwired for now
  
  
  int threadsPerBlock = min(maxThreadsPerBlock, nRandNums);
  
  int xBlocks = 0;
  int yBlocks = 0;
  
  if( nRandNums > maxThreadsPerBlock){
    int nBlocks = RoundUp(  nRandNums/sqrt(maxThreadsPerBlock) );
    xBlocks = RoundUp(  nRandNums/xMaxThreadsPerBlock );
    yBlocks = RoundUp(  nRandNums/yMaxThreadsPerBlock );
  }else{
    xBlocks = 1;   // if matrix is smaller than 1 block
    yBlocks = 1;
  }
  
  int nBlocks = xBlocks = yBlocks;           // Assumption that
  int me = xBlocks * yBlocks * threadsPerBlock;
  
  fprintf(stdout, "    xBlocks: %d, yBlocks: %d, nRandNums: %d BLKWIDTH: %d, threadsPerBlock %d ",xBlocks, yBlocks, nRandNums, BLKWIDTH, threadsPerBlock);
  fprintf(stdout, "    number of threads: %d\n",me);
  
  //__________________________________
  //  Kernel invocation
  dim3 dimBlock(BLKWIDTH, BLKWIDTH, 1);
  dim3 dimGrid( xBlocks,  yBlocks,  1);
  
  
  // setup random number generator states on the device, 1 for each thread
  hiprandState* randNumStates;
  int numStates = dimGrid.x * dimGrid.y * dimBlock.x * dimBlock.y * dimBlock.z;
  hipMalloc((void**)&randNumStates, numStates * sizeof(hiprandState));
  
  //__________________________________
  //  Global Memory Kernel
  randNumKernel<<<dimGrid, dimBlock>>>( randNumStates, Md, Nd, nRandNums );
  
  //__________________________________
  //   copy from device memory and free memory
  hipMemcpy( M, Md, size, hipMemcpyDeviceToHost );
  hipMemcpy( N, Nd, size, hipMemcpyDeviceToHost );
  hipFree( Md );
  hipFree( Nd );
  hipFree(randNumStates) ;
}


//______________________________________________________________________
int main( int argc, char** argv)
{  

//  for(int power = 4; power<8; ++power) { 
//    int nRandNums = pow(10,power);
    int nRandNums = 10000;   
    fprintf(stdout,"__________________________________\n");
    fprintf(stdout," nRand %d  \n", nRandNums);
    
    //__________________________________
    //  allocate memory
    unsigned int size = nRandNums;
    unsigned int mem_size = sizeof(double) * size;
    double* rand_CPU       = (double*)malloc(mem_size); 
    double* rand_hostGPU   = (double*)malloc(mem_size);
    double* rand_devGPU_M  = (double*)malloc(mem_size);
    double* rand_devGPU_N  = (double*)malloc(mem_size); 
       
    time_t start;
    start = time(NULL);
    //__________________________________
    //  Compute the random numbers
    randCPU( rand_CPU, nRandNums );
    stopwatch(" randCPU: ", start);
    
    start = time(NULL);
    randHostGPU( rand_hostGPU, nRandNums);
    stopwatch(" randHostGPU: ", start);
     
    start = time(NULL);    
    randDeviceGPU( rand_devGPU_M, rand_devGPU_N, nRandNums);
    stopwatch(" randHostGPU: ", start);
    
    //__________________________________
    //  Output data
    FILE *fp;
    fp = fopen("randomNumbers.dat", "w");
    
    for (int i = 0; i< nRandNums; i++){
      fprintf( fp, "%i, %16.15E, %16.15E, %16.15E,  %16.15E\n",i, rand_CPU[i], rand_hostGPU[i], rand_devGPU_M[i], rand_devGPU_N[i] );
    }
    fclose(fp);
    
    //__________________________________
    //Free memory
    free( rand_CPU );
    free( rand_hostGPU );
    free( rand_devGPU_M );
    free( rand_devGPU_N );
//  }   // loop 
}




